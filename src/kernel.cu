
#include "hip/hip_runtime.h"


#include <stdio.h>
//#include <unistd.h>
#include <vector>
#include <iostream>
using namespace std;

#ifndef __CUDACC__  
    #define __CUDACC__
#endif


__device__ int minIndex(double* data,double* centers,bool* centerflags,int kCenter,int dimension)
{
	int result;
	int firstindex=0;

	while(firstindex<kCenter && (! centerflags[ firstindex]))
		++firstindex;

	result=firstindex;
	double mindis=0.0;

	for (int i = 0; i < dimension; i++)
	{
		double tdis=centers[firstindex*dimension+i]-data[i];
		mindis+=tdis*tdis;
	}

	for(int i=firstindex+1;i<kCenter;++i)
	{
		if(centerflags[i])
		{
			double tdis=0.0;
			for (int j = 0; j < dimension; j++)
			{
				double ttdis=centers[i*dimension+j]-data[j];
				tdis+=ttdis*ttdis;
			}
			if(tdis<mindis)
			{
				mindis=tdis;
				result=i;
			}
		}
	}

	return result;
}
vector<int> shuffledOrder(int n,int m)
{
	//assert(n>=m);
	vector<int> result(m);
	vector<int> index(n);
	for(int i=0;i<n;++i)
	{
		index[i]=i;
	}

	for(int i=0;i<m;++i)
	{
		int tem=rand()%(n-i);
		result[i]=index[tem];
		index[tem]=index[n-i-1];

	}
	return result;
}
__global__ void updatebelonging4(int index,double* dataset,int datasize,int dimension,double* centers,int* labels,int kCenter,bool* goodCenterFlag,bool* CenterChangeFlag)
{
	int j=index+blockDim.x* blockIdx.x +threadIdx.x;

	int tlabel;
	if(j<datasize)
	{
		tlabel=minIndex(dataset+j*dimension, centers,goodCenterFlag,kCenter,dimension);
		if(tlabel!=labels[j])
			CenterChangeFlag[j]=true;
		labels[j]=tlabel;

	}
	
	
}

__global__ void updateCorresponds(int* labels,int datasize,int kCenter,int* correspondings,bool* centerChangeFlag,int* centerStartIndex,int* centerCount,int* curCount,bool* goodCenterFlag,bool* nochange)
{
//	cudaMemset(correspondings,0,sizeof(int)*datasize);
//	cudaMemset(centerCount,0,sizeof(int)*kCenter);
	for (int i = 0; i < kCenter; i++)
	{
		centerCount[i]=0;
		curCount[i]=0;
	}
	for (int i = 0; i < datasize; i++)
	{
		++centerCount[labels[i]];

		if(centerChangeFlag[i])
		{
			*nochange=false;
			centerChangeFlag[i]=false;
		}	
	}
	centerStartIndex[0]=0;
	for (int i = 0; i < kCenter; i++)
	{
		if (centerCount[i]==0)
		{
			goodCenterFlag[i]=false;
		}
		if(i>0)
		{
			centerStartIndex[i]=centerStartIndex[i-1]+centerCount[i-1];
		}
	}
	
	//curCount=new int[kCenter];

	for (int i = 0; i < datasize; i++)
	{
		int tlabel=labels[i];
		//int ind=;
		correspondings[centerStartIndex[tlabel] + curCount[tlabel]]=i;
		++curCount[tlabel];
	}

	


}

__global__ void updateCenters4(int ind,double* dataset,int datasize,int dimension,double* centers,int kCenter,int* corresponding,int* centerStartIndex,int* centerCount)
{
	int j=ind+blockDim.x*blockIdx.x+threadIdx.x;
	if(j<kCenter)
	{
		if(centerCount[j]>0)
		{
			for (int i = 0; i < dimension; i++)
			{
				centers[j*dimension+i]=0;
			}
			for (int i = 0; i < centerCount[j]; i++)
			{
				int curinde=corresponding[ centerStartIndex[j]+i];

				for (int k = 0; k < dimension; k++)
				{
					centers[j*dimension+k]+=dataset[curinde*dimension+k]/centerCount[j];
				}
			}

		}
	}
}

void kmeans4(double* dataset,int datasize,int dimension,double* centers,int* labels,int kCenter,int maxIterationNumber,int threadsize,int blocksize=65535)
{

	vector<int> initialCenterIndex=shuffledOrder(datasize,kCenter);

	for(int i=0;i<kCenter;++i)
		for(int j=0;j<dimension;++j)
			centers[i*dimension+j]=dataset[initialCenterIndex[i]*dimension+j];


	bool* goodCenterFlag;
	int* centerCount;
	int* curCount;
	bool* centerChangeFlag;
	int* corresponding;
	int* centerStartIndex;

	bool* noChange;
	

	hipMallocManaged(&goodCenterFlag,sizeof(bool)*kCenter);

	hipMallocManaged(&centerCount,sizeof(int)*kCenter);
	hipMallocManaged(&curCount,sizeof(int)*kCenter);

	hipMallocManaged(&centerChangeFlag,sizeof(bool)*datasize);

	hipMallocManaged(&corresponding,sizeof(bool)*datasize);

	hipMallocManaged(&centerStartIndex,sizeof(int)*kCenter);

	hipMalloc(&noChange,sizeof(bool));
	
	for(int i=0;i<kCenter;++i) goodCenterFlag[i]=true;

	for(int i=0;i<kCenter;++i) centerCount[i]=0;

	for(int i=0;i<datasize;++i) centerChangeFlag[i]=false;

	hipError_t error;

	for(int iterN=0;iterN<maxIterationNumber;++iterN)
	{

		int remain=datasize;
		while(remain>0)
		{
			int tblocksize=blocksize;
			if(blocksize*threadsize>=remain)
			{
				tblocksize=remain/threadsize+(remain%threadsize==0?0:1);
			}

			updatebelonging4<<<tblocksize,threadsize>>>(datasize-remain,dataset,datasize,
				dimension,centers,labels,
				kCenter,goodCenterFlag,
				centerChangeFlag);

			remain-=tblocksize*threadsize;
		}
		//printf("belongings ok\n");

		error = hipGetLastError();
		if(error != hipSuccess)
		{

			printf("belong CUDA error: %s\n", hipGetErrorString(error));

		}
		updateCorresponds<<<1,1>>>(labels,datasize,kCenter,corresponding,centerChangeFlag,centerStartIndex,centerCount,curCount,goodCenterFlag,noChange);

		error = hipGetLastError();
		if(error != hipSuccess)
		{
		// print the CUDA error message and exit
			printf("corresponding  CUDA error: %s\n", hipGetErrorString(error));
		// exit(-1);
		}
		remain=kCenter;

		while(remain>0)
		{
			//printf("%d ",i);
			int tblocksize=blocksize;
			if(blocksize*threadsize>remain)
			{
				tblocksize=remain/threadsize+(remain%threadsize==0?0:1);
			}

			updateCenters4<<<tblocksize,threadsize>>>(kCenter-remain,dataset,datasize,dimension,centers,kCenter,corresponding,centerStartIndex,centerCount);
			remain-=tblocksize*threadsize;
		}

		//printf("center ok\n");
		
		error = hipGetLastError();
		if(error != hipSuccess)
		{
		// print the CUDA error message and exit
			printf("center update CUDA error: %s\n", hipGetErrorString(error));
		// exit(-1);
		}

		printf("finished iteration NO. %d\n",iterN);

		bool hnochange;
		hipMemcpy(&hnochange,noChange,sizeof(bool),hipMemcpyDeviceToHost);

		error = hipGetLastError();
		if(error != hipSuccess)
		{
		// print the CUDA error message and exit
			printf("memcopy free CUDA error: %s\n", hipGetErrorString(error));
		// exit(-1);
		}
		if(hnochange)
			break;

	}



	hipError_t cudaStatus;

	error = hipGetLastError();
	if(error != hipSuccess)
	{
	// print the CUDA error message and exit
		printf("before free CUDA error: %s\n", hipGetErrorString(error));
	// exit(-1);
	}


	hipFree(noChange);
	hipFree(goodCenterFlag);

	hipFree(corresponding);
	hipFree(centerStartIndex);

	hipFree(centerCount);
	hipFree(curCount);
	hipFree(centerChangeFlag);
	hipFree(dataset);
	error = hipGetLastError();
	if(error != hipSuccess)
	{
	// print the CUDA error message and exit
		printf("before CUDA error: %s\n", hipGetErrorString(error));
	// exit(-1);
	}

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
	// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
	// exit(-1);
	}


}




int main()
{
	//chdir("/home/pitybea/");

	hipDeviceProp_t prop;

	hipGetDeviceProperties(&prop,0);
	cout<<prop.maxThreadsPerBlock<<endl;


	double* dataset;
	FILE* fp=fopen("fea.txt","r");

	int size,dimension;

	fscanf(fp,"%d %d\n",&size,&dimension);

	size=300000;

	hipMallocManaged(&dataset,sizeof(double)*size*dimension);
	printf("%d %d\n",size,dimension);

	for (int i=0;i<size;i++)
	{
		if(i%10000==0) printf("%d\t",i);
		for (int j=0;j<dimension;j++)
		{
			fscanf(fp,"%lf ",&dataset[i*dimension+j]);
		}
		fscanf(fp,"\n");
	}

	fclose(fp);



	int k=size/1000;
	double* centers;
	int* labels;

	hipMallocManaged(&centers,sizeof(double)*k*dimension);
	hipMallocManaged(&labels,sizeof(int)*size);

	for(int i=0;i<k*dimension;++i)
		centers[i]=0;
	for(int i=0;i<size;++i)
		labels[i]=0;

	kmeans4(dataset,size,dimension,centers,labels,k,12,prop.maxThreadsPerBlock);

	cout<<labels[0]<<" "<<endl;
	//FILE* fp;
	fp=fopen("labels.txt","w");
	fprintf(fp,"%d\n",size);
	for(int i=0;i<size;i++)
	{
		if(i%1000==0)
		cout<<labels[i]<<" ";
		fprintf(fp,"%d\n",labels[i]);
	}

	fclose(fp);
	/*
	fp=fopen("centers.txt","w");
	fprintf(fp,"%d %d\n",k,dimension);
	for(int i=0;i<k;i++)
	{
		for(int j=0;j<dimension;j++)
			fprintf(fp,"%lf ",centers[i*dimension+j]);

		fprintf(fp,"\n");
	}
	fclose(fp);
	 */
	hipFree(labels);
	hipFree(centers);

	hipDeviceReset();
	return 0;
}



