#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

static const int WORK_SIZE = 256;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__host__ __device__ unsigned int bitreverse(unsigned int number) {
	number = ((0xf0f0f0f0 & number) >> 4) | ((0x0f0f0f0f & number) << 4);
	number = ((0xcccccccc & number) >> 2) | ((0x33333333 & number) << 2);
	number = ((0xaaaaaaaa & number) >> 1) | ((0x55555555 & number) << 1);
	return number;
}

/**
 * CUDA kernel function that reverses the order of bits in each element of the array.
 */

__global__ void adda(int count,double* a,double *b,double* c,int st)
{

	int i= st+blockDim.x* blockIdx.x +threadIdx.x;
	if(i<count)
		c[i]=a[i]+b[i];
}

__global__ void mm(int count,double* c,int st)
{

	int i= st+blockDim.x* blockIdx.x +threadIdx.x;
	if(i<count)
		c[i]/=2;
}

void launch(double* a,double* b,double* c,int testsize)
{
	int threadsize=256;
	int blocksize=256;
	for(int i=0;i<testsize;i+=threadsize*blocksize)
		adda<<<blocksize,threadsize>>>(testsize,a,b,c,i);

	for(int i=0;i<testsize;i+=threadsize*blocksize)
		mm<<<blocksize,threadsize>>>(testsize,c,i);

	hipDeviceSynchronize();
}

int helloworld()
{
	int testsize=10000000;
	double* a;
	double* b;



	hipMallocManaged(&a,sizeof(double)*testsize);
	hipMallocManaged(&b,sizeof(double)*testsize);

	for (int i = 0; i < testsize; i++)
	{
		a[i]=i;
		b[i]=testsize-i;
	}
	double* c;


	hipMallocManaged(&c,sizeof(double)*testsize);
	launch(a,b,c,testsize);

	for (int i = 0; i < testsize; i++)
	{

		if(i%1000000==0)
			printf("%f ",c[i]);

	}

	hipFree(a);
	hipFree(b);
	hipFree(c);

	hipDeviceReset();
    return 0;
}




__global__ void updatebelonging(int index,double* dataset,int datasize,int dimension,double* centers,int* labels,int kCenter,bool* goodCenterFlag,int* paraClusterCount,double* paraCenters,bool* paraCenterChangeFlag)
{
	int j=index+blockDim.x* blockIdx.x +threadIdx.x;
	int pind=blockDim.x* blockIdx.x +threadIdx.x;
	if(j<datasize)
	{
		int tlabel;
		int firstindex=0;
		while(firstindex<kCenter && (!goodCenterFlag[firstindex]))
			++firstindex;

		tlabel=firstindex;
		double minDis=0.0;

		for(int i=0;i<dimension;++i)
		{
			double tdis=dataset[j*dimension+i]-centers[firstindex*dimension+i];
			minDis+=tdis*tdis;
		}
		for(int k=firstindex+1;k<kCenter;++k)
		{
			if(goodCenterFlag[k])
			{
				double curdis=0.0;
				for(int i=0;i<dimension;++i)
				{
					double tdis=dataset[j*dimension+i]-centers[k*dimension+i];
					curdis+=tdis*tdis;
				}
				if(curdis<minDis)
				{
					minDis=curdis;
					tlabel=k;
				}
			}
		}
		if(tlabel!=labels[j])
			paraCenterChangeFlag[blockDim.x* blockIdx.x +threadIdx.x]=true;

		labels[j]=tlabel;
		++paraClusterCount[pind*kCenter+tlabel];
		for(int k=0;k<dimension;++k)
		{
			paraCenters[pind*(kCenter*dimension)+tlabel*dimension+k]+=dataset[j*dimension+k];
		}
	}

}
__global__ void updateCenter(int index,int dimension,int parallelNumber,double* centers,int kCenter,bool* goodCenterFlag,int* paraClusterCount,double* paraCenters,int* clusterCount)
{
	int j=index+blockDim.x* blockIdx.x +threadIdx.x;
	if(j<kCenter)
	{
		for(int i=0;i<dimension;++i)
			centers[j*dimension+i]=0.0;

		clusterCount[j]=0;

		for(int i=0;i<parallelNumber;++i)
		{
			for(int l=0;l<dimension;++l)
			{
				centers[j*dimension+l]+=paraCenters[i*kCenter*dimension+j*dimension+l];
				paraCenters[i*kCenter*dimension+j*dimension+l]=0.0;
			}
			clusterCount[j]+=paraClusterCount[i*kCenter+j];
			paraClusterCount[i*kCenter+j]=0;
		}
		if(clusterCount[j]==0)
			goodCenterFlag[j]=false;
		else
		{
			for(int i=0;i<dimension;++i)
				centers[j*dimension+i]/=clusterCount[j];
		}
	}

}
void kmeans(double* dataset,int datasize,int dimension,double* centers,int* labels,int kCenter,int maxIterationNumber)
{
	int threadsize=32;
	int blocksize=32;
	vector<int> initialCenterIndex=shuffledOrder(datasize,kCenter);

	for(int i=0;i<kCenter;++i)
		for(int j=0;j<dimension;++j)
			centers[i*dimension+j]=dataset[initialCenterIndex[i]*dimension+j];

	/*
	 vector<bool> goodCenterFlag(kCenter,true);
	vector<vector<int> > paraClusterCount(parallelNumber,vector<int>(kCenter,0));
	vector<int> clusterCount(kCenter,0);

	vector<vector<vector<double> > > paraCenters(parallelNumber,vector<vector<double> >(kCenter,vector<double>(dataset[0].size(),0.0)));

//	vector<bool> centerChangeFlag(dataset.size(),false);
	vector<bool> paraCenterChangeFlag(parallelNumber,false);
	 */

	int parallelNumber=threadsize*blocksize;
	bool* goodCenterFlag;
	int* paraClusterCount;
	int* clusterCount;
	double* paraCenters;
	bool* paraCenterChangeFlag;

	hipError_t cudaStatus;
	hipMallocManaged(&goodCenterFlag,sizeof(bool)*kCenter);
	printf("\ns1\n");
	hipMallocManaged(&paraClusterCount,sizeof(int)*parallelNumber*kCenter);
	printf("s2\n");
	hipMallocManaged(&clusterCount,sizeof(int)*kCenter);
	printf("s3\n");
	cudaStatus=hipMallocManaged(&paraCenters,sizeof(double)*dimension*kCenter*parallelNumber);


	printf("s4\n");
	hipMallocManaged(&paraCenterChangeFlag,sizeof(bool)*parallelNumber);
	printf("s5\n");

	for(int i=0;i<kCenter;++i) goodCenterFlag[i]=true;
	printf("\ns1\n");
	for(int i=0;i<parallelNumber*kCenter;++i) paraClusterCount[i]=0;
	printf("s2\n");
	for(int i=0;i<kCenter;++i) clusterCount[i]=0;
	printf("s3\n");
	for(int i=0;i<dimension*kCenter*parallelNumber;++i)
	{
		paraCenters[i]=0.0;
	}
	printf("s4\n");
	for(int i=0;i<parallelNumber;++i) paraCenterChangeFlag[i]=false;
	printf("s5\n");

	for(int iterN=0;iterN<maxIterationNumber;++iterN)
	{
		for(int i=0;i<datasize;i+=parallelNumber)
		{
			//printf("%d ",i);
			updatebelonging<<<blocksize,threadsize>>>(i,dataset,datasize,dimension,centers,labels,kCenter,goodCenterFlag,paraClusterCount,paraCenters,paraCenterChangeFlag);
		}
		printf("belongings ok\n");

		for(int i=0;i<kCenter;i+=parallelNumber)
		{
			updateCenter<<<blocksize,threadsize>>>(i,dimension,parallelNumber,centers,kCenter,goodCenterFlag,paraClusterCount,paraCenters,clusterCount);
		}
		printf("center ok\n");
		//hipDeviceSynchronize();
		/*bool noChange=true;

		for(int i=0;i<parallelNumber;i++)
		{
			if(paraCenterChangeFlag[i]==true)
			{
				noChange=false;
				paraCenterChangeFlag[i]=false;
			}
		}
		if(noChange)
			break;*/
		printf("finished iteration NO. %d\n",iterN);

	}

	hipDeviceSynchronize();
	hipFree(goodCenterFlag);
	printf("s7\n");
	hipFree(paraClusterCount);
	printf("s8\n");
	hipFree(clusterCount);
	printf("s9\n");
	hipFree(paraCenters);
	printf("s10\n");
	hipFree(paraCenterChangeFlag);
	printf("s11\n");
}


__global__ void updatebelonging2(int index,double* dataset,int datasize,int dimension,double* centers,int* labels,int kCenter,bool* goodCenterFlag,int* paraClusterCount,bool* paraCenterChangeFlag)
{
	int j=index+blockDim.x* blockIdx.x +threadIdx.x;
	int pind=blockDim.x* blockIdx.x +threadIdx.x;
	if(j<datasize)
	{
		int tlabel;
		int firstindex=0;
		while(firstindex<kCenter && (!goodCenterFlag[firstindex]))
			++firstindex;

		tlabel=firstindex;
		double minDis=0.0;

		for(int i=0;i<dimension;++i)
		{
			double tdis=dataset[j*dimension+i]-centers[firstindex*dimension+i];
			minDis+=tdis*tdis;
		}
		for(int k=firstindex+1;k<kCenter;++k)
		{
			if(goodCenterFlag[k])
			{
				double curdis=0.0;
				for(int i=0;i<dimension;++i)
				{
					double tdis=dataset[j*dimension+i]-centers[k*dimension+i];
					curdis+=tdis*tdis;
				}
				if(curdis<minDis)
				{
					minDis=curdis;
					tlabel=k;
				}
			}
		}
		if(tlabel!=labels[j])
			paraCenterChangeFlag[blockDim.x* blockIdx.x +threadIdx.x]=true;

		labels[j]=tlabel;
		++paraClusterCount[pind*kCenter+tlabel];
		for(int k=0;k<dimension;++k)
		{
//			paraCenters[pind*(kCenter*dimension)+tlabel*dimension+k]+=dataset[j*dimension+k];
		}
	}

}
__global__ void updateCenter2(int index,int dimension,int parallelNumber,double* centers,int kCenter,bool* goodCenterFlag,int* paraClusterCount,int* clusterCount)
{
	int j=index+blockDim.x* blockIdx.x +threadIdx.x;
	if(j<kCenter)
	{
		for(int i=0;i<dimension;++i)
			centers[j*dimension+i]=0.0;

		clusterCount[j]=0;

		for(int i=0;i<parallelNumber;++i)
		{
			for(int l=0;l<dimension;++l)
			{
				//centers[j*dimension+l]+=paraCenters[i*kCenter*dimension+j*dimension+l];
				//paraCenters[i*kCenter*dimension+j*dimension+l]=0.0;
			}
			clusterCount[j]+=paraClusterCount[i*kCenter+j];
			paraClusterCount[i*kCenter+j]=0;
		}
		if(clusterCount[j]==0)
			goodCenterFlag[j]=false;
		else
		{
			for(int i=0;i<dimension;++i)
				centers[j*dimension+i]/=clusterCount[j];
		}
	}

}
void kmeans2(double* dataset,int datasize,int dimension,double* centers,int* labels,int kCenter,int maxIterationNumber)
{
	int threadsize=256;
	int blocksize=256;
	vector<int> initialCenterIndex=shuffledOrder(datasize,kCenter);

	for(int i=0;i<kCenter;++i)
		for(int j=0;j<dimension;++j)
			centers[i*dimension+j]=dataset[initialCenterIndex[i]*dimension+j];

	/*
	 vector<bool> goodCenterFlag(kCenter,true);
	vector<vector<int> > paraClusterCount(parallelNumber,vector<int>(kCenter,0));
	vector<int> clusterCount(kCenter,0);

	vector<vector<vector<double> > > paraCenters(parallelNumber,vector<vector<double> >(kCenter,vector<double>(dataset[0].size(),0.0)));

//	vector<bool> centerChangeFlag(dataset.size(),false);
	vector<bool> paraCenterChangeFlag(parallelNumber,false);
	 */

	int parallelNumber=threadsize*blocksize;
	bool* goodCenterFlag;
	int* paraClusterCount;
	int* clusterCount;
	//double* paraCenters;
	bool* paraCenterChangeFlag;

	hipError_t cudaStatus;
	hipMallocManaged(&goodCenterFlag,sizeof(bool)*kCenter);
	printf("\ns1\n");
	hipMallocManaged(&paraClusterCount,sizeof(int)*parallelNumber*kCenter);
	printf("s2\n");
	hipMallocManaged(&clusterCount,sizeof(int)*kCenter);
	printf("s3\n");
	//cudaStatus=hipMallocManaged(&paraCenters,sizeof(double)*dimension*kCenter*parallelNumber);


	printf("s4\n");
	hipMallocManaged(&paraCenterChangeFlag,sizeof(bool)*parallelNumber);
	printf("s5\n");

	for(int i=0;i<kCenter;++i) goodCenterFlag[i]=true;
	printf("\ns1\n");
	for(int i=0;i<parallelNumber*kCenter;++i) paraClusterCount[i]=0;
	printf("s2\n");
	for(int i=0;i<kCenter;++i) clusterCount[i]=0;
	printf("s3\n");

	printf("s4\n");
	for(int i=0;i<parallelNumber;++i) paraCenterChangeFlag[i]=false;
	printf("s5\n");

	for(int iterN=0;iterN<maxIterationNumber;++iterN)
	{
		for(int i=0;i<datasize;i+=parallelNumber)
		{
			//printf("%d ",i);
			updatebelonging2<<<blocksize,threadsize>>>(i,dataset,datasize,dimension,centers,labels,kCenter,goodCenterFlag,paraClusterCount,paraCenterChangeFlag);
		}
		printf("belongings ok\n");

		for(int i=0;i<kCenter;i+=parallelNumber)
		{
			updateCenter2<<<blocksize,threadsize>>>(i,dimension,parallelNumber,centers,kCenter,goodCenterFlag,paraClusterCount,clusterCount);
		}
		printf("center ok\n");
		//hipDeviceSynchronize();
		/*bool noChange=true;

		for(int i=0;i<parallelNumber;i++)
		{
			if(paraCenterChangeFlag[i]==true)
			{
				noChange=false;
				paraCenterChangeFlag[i]=false;
			}
		}
		if(noChange)
			break;*/
		printf("finished iteration NO. %d\n",iterN);

	}

	hipDeviceSynchronize();
	hipFree(goodCenterFlag);
	printf("s7\n");
	hipFree(paraClusterCount);
	printf("s8\n");
	hipFree(clusterCount);
	printf("s9\n");

	printf("s10\n");
	hipFree(paraCenterChangeFlag);
	printf("s11\n");
}



//this is not going to work, there are not enough of shared memory
__global__ void updatebelonging3(int index,double* dataset,int datasize,int dimension,double* centers,int* labels,int kCenter,bool* goodCenterFlag,double* blockCenters,bool* CenterChangeFlag)
{
	int j=index+blockDim.x* blockIdx.x +threadIdx.x;

	extern __shared__ double localBlockCenters[];
	//int pind=blockDim.x* blockIdx.x +threadIdx.x;

	for (int i = 0; i < dimension*kCenter; i++)
	{
		localBlockCenters[threadIdx.x*(dimension*kCenter)+ i]=0.0;
	}
	int tlabel;

	if(j<datasize)
	{
		tlabel=minIndex(dataset+j*dimension, centers,goodCenterFlag,kCenter,dimension);

		if(tlabel!=labels[j])
			CenterChangeFlag[j]=true;

		labels[j]=tlabel;
		for (int i = 0; i < dimension; i++)
		{
			localBlockCenters[threadIdx.x*(dimension*kCenter)+ tlabel* dimension+i]+=dataset[j*dimension+i];
		}

	}
	__syncthreads();
	for (int size = blockDim.x; size >1 ; size=(size+1)/2)
	{
		int offset=size/2;
		if(threadIdx.x<offset)
		for (int i = 0; i < dimension*kCenter; i++)
		{
			localBlockCenters[threadIdx.x*(dimension*kCenter)+ i]+=localBlockCenters[(size-1-threadIdx.x)*(dimension*kCenter)+ i];
		}
		__syncthreads();
	}

	if(threadIdx.x==0)
	{
		for (int i = 0; i < dimension*kCenter; i++)
		{
			blockCenters[blockIdx.x*dimension*kCenter+i]=localBlockCenters[i];
		}
	}
}



__global__ void updateCenter3(int index,int dimension,int parallelNumber,double* centers,int kCenter,bool* goodCenterFlag,int* blockCenterCount,int* centerCount)
{
	int j=index+blockDim.x* blockIdx.x +threadIdx.x;

	if(j<kCenter)
	{
		for(int i=0;i<dimension;++i)
			centers[j*dimension+i]=0.0;

		centerCount[j]=0;

		for(int i=0;i<parallelNumber;++i)
		{
			for(int l=0;l<dimension;++l)
			{
				//centers[j*dimension+l]+=paraCenters[i*kCenter*dimension+j*dimension+l];
				//paraCenters[i*kCenter*dimension+j*dimension+l]=0.0;
			}
			//clusterCount[j]+=paraClusterCount[i*kCenter+j];
		//	paraClusterCount[i*kCenter+j]=0;
		}

	}

}
